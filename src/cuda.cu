#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2025 Guilherme Leoi <leoi.guilherme@aluno.ufabc.edu.br>
//
// SPDX-License-Identifier: AGPL-3.0-only

#include "filter.h"
#include "ppm.h"
#include <stddef.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

// PPM SECTION

#define ASSERT(expr, msg, exit_label)                                          \
  if (!(expr)) {                                                               \
    puts(msg);                                                                 \
    goto exit_label;                                                           \
  }
#define MAX_LINE 4096

PpmImage *read_ppm_image(FILE *source_file) {
  PpmImage *image = NULL;
  size_t image_size = 0;
  ASSERT(source_file != NULL, "Source file is NULL", read_ppm_image_error);
  image = (PpmImage*)malloc(sizeof(PpmImage));
  char header[2];
  ASSERT(fscanf(source_file, "%c%c", &header[0], &header[1]),
         "Error reading the file header", read_ppm_image_error);
  ASSERT(header[0] == 'P' && header[1] == '3',
         "Unsupported format (expected `P3`)", read_ppm_image_error)
  char line[MAX_LINE];
  do {
    ASSERT(fgets(line, MAX_LINE, source_file),
           "Error reading line(s) after header", read_ppm_image_error);
  } while (line[0] == '#' || line[0] == '\n');
  ASSERT(sscanf(line, "%lu %lu", &image->width, &image->height),
         "Error reading `width` and `height` integers", read_ppm_image_error);
  ASSERT(fscanf(source_file, "%hu", &image->max_value),
         "Error reading `max_value` integer", read_ppm_image_error);
  image_size = image->width * image->height;
  image->color_values_write = (RgbTriplet*)malloc(image_size * sizeof(RgbTriplet));
  image->color_values_read = (RgbTriplet*)malloc(image_size * sizeof(RgbTriplet));
  image->needs_flushing = 0;
  for (size_t idx = 0; idx < image_size; idx++) {
    uint16_t red, green, blue;
    ASSERT(fscanf(source_file, "%hu %hu %hu", &red, &green, &blue),
           "Error reading `red`, `blue` and `green` integers",
           read_ppm_image_error);
    RgbTriplet rgb =
        (RgbTriplet){.r = ((float)red) / ((float)image->max_value),
                     .g = ((float)green) / ((float)image->max_value),
                     .b = ((float)blue) / ((float)image->max_value)};
    write_at_idx_ppm_image(image, idx, rgb);
  }
  ASSERT(flush_ppm_image(image), "Error flushing the image write buffer",
         read_ppm_image_error);
  return image;
read_ppm_image_error:
  free_ppm_image(&image);
  return NULL;
}

int write_at_idx_ppm_image(PpmImage *image, size_t idx, RgbTriplet rgb) {
  ASSERT(image != NULL, "PPM image is NULL", write_at_idx_ppm_image_error);
  ASSERT(image->color_values_write != NULL, "PPM image write buffer is NULL",
         write_at_idx_ppm_image_error);
  ASSERT(idx < (image->width * image->height),
         "Error writing at out of bounds index from PPM image",
         write_at_idx_ppm_image_error);
  image->color_values_write[idx] = rgb;
  image->needs_flushing = 1;
  return 1;
write_at_idx_ppm_image_error:
  return 0;
}

int write_at_xy_ppm_image(PpmImage *image, size_t x, size_t y, RgbTriplet rgb) {
  ASSERT(image != NULL, "PPM image is NULL", write_at_xy_ppm_image_error);
  return write_at_idx_ppm_image(image, x + y * image->width, rgb);
write_at_xy_ppm_image_error:
  return 0;
}

int flush_ppm_image(PpmImage *image) {
  ASSERT(image != NULL, "PPM image is NULL", flush_ppm_image_error);
  ASSERT(image->color_values_read != NULL, "PPM image read buffer is NULL",
         flush_ppm_image_error);
  ASSERT(image->color_values_write != NULL, "PPM image write buffer is NULL",
         flush_ppm_image_error);
  if (image->needs_flushing) {
    size_t image_size = image->width * image->height;
    for (size_t idx = 0; idx < image_size; idx++)
      image->color_values_read[idx] = image->color_values_write[idx];
    image->needs_flushing = 0;
  }
  return 1;
flush_ppm_image_error:
  return 0;
}

int read_at_idx_ppm_image(PpmImage *image, size_t idx, RgbTriplet *rgb) {
  ASSERT(image != NULL, "PPM image is NULL", read_at_idx_ppm_image_error);
  ASSERT(image->color_values_read != NULL, "PPM image read buffer is NULL",
         read_at_idx_ppm_image_error);
  ASSERT(rgb != NULL, "RgbTriplet is NULL", read_at_idx_ppm_image_error);
  ASSERT(idx < (image->width * image->height),
         "Error reading at out of bounds index from PPM image",
         read_at_idx_ppm_image_error);
  *rgb = image->color_values_read[idx];
  return 1;
read_at_idx_ppm_image_error:
  return 0;
}

int read_at_xy_ppm_image(PpmImage *image, size_t x, size_t y, RgbTriplet *rgb) {
  ASSERT(image != NULL, "PPM image is NULL", read_at_xy_ppm_image_error);
  return read_at_idx_ppm_image(image, x + y * image->width, rgb);
read_at_xy_ppm_image_error:
  return 0;
}

int save_ppm_image(PpmImage *image, FILE *output_file) {
  size_t image_size = 0;
  ASSERT(image != NULL, "PPM image is NULL", save_ppm_image_error);
  ASSERT(output_file != NULL, "Output file is NULL", save_ppm_image_error);
  ASSERT(fprintf(output_file, "P3\n"), "Error writing PPM image header",
         save_ppm_image_error);
  ASSERT(fprintf(output_file, "%lu %lu\n", image->width, image->height),
         "Error writing `width` and `height` integers", save_ppm_image_error);
  ASSERT(fprintf(output_file, "%hu\n", image->max_value),
         "Error writing `max_value` integer", save_ppm_image_error);
  image_size = image->width * image->height;
  for (size_t idx = 0; idx < image_size; idx++) {
    RgbTriplet rgb;
    ASSERT(read_at_idx_ppm_image(image, idx, &rgb),
           "Error reading at index from PPM image", save_ppm_image_error);
    uint16_t red, green, blue;
    red = (uint16_t)roundf(rgb.r * ((float)image->max_value));
    green = (uint16_t)roundf(rgb.g * ((float)image->max_value));
    blue = (uint16_t)roundf(rgb.b * ((float)image->max_value));
    ASSERT(fprintf(output_file, "%hu %hu %hu\n", red, green, blue),
           "Error writing `red`, `green` and `blue` integers",
           save_ppm_image_error);
  }
  return 1;
save_ppm_image_error:
  return 0;
}

void free_ppm_image(PpmImage **image) {
  if (image == NULL || *image == NULL)
    return;
  if ((*image)->color_values_write) {
    free((*image)->color_values_write);
    (*image)->color_values_write = NULL;
  }
  if ((*image)->color_values_read) {
    free((*image)->color_values_read);
    (*image)->color_values_read = NULL;
  }
  free(*image);
  *image = NULL;
}

// FILTER SECTION

__device__ void r_pixel(PpmImage *image, size_t m, size_t x, size_t y, size_t *radius) {
  RgbTriplet rgb;
  size_t idx = x + y * image->width;
  rgb = image->color_values_read[idx];
  float sum = rgb.r + rgb.g + rgb.b;
  *radius = (((size_t)(sum * 255)) % m) + 1;
}

__device__ void blur_at(PpmImage *image, size_t m, size_t x, size_t y, RgbTriplet *rgb) {
  size_t radius;
  r_pixel(image, m, x, y, &radius);
  float sum_r = 0.0f, sum_g = 0.0f, sum_b = 0.0f;
  for (size_t i = 0; i <= 2 * radius; i++) {
    for (size_t j = 0; j <= 2 * radius; j++) {
      size_t neighbour_x = x + i;
      if (neighbour_x < radius)
        neighbour_x = 0;
      else
        neighbour_x -= radius;
      size_t neighbour_y = y + j;
      if (neighbour_y < radius)
        neighbour_y = 0;
      else
        neighbour_y -= radius;
      if (neighbour_x >= image->width)
        neighbour_x = image->width - 1;
      if (neighbour_y >= image->height)
        neighbour_y = image->height - 1;
      size_t neighbour_idx = neighbour_x + neighbour_y * image->width;
      RgbTriplet neighbour_rgb;
      neighbour_rgb = image->color_values_read[neighbour_idx];
      sum_r += neighbour_rgb.r;
      sum_g += neighbour_rgb.g;
      sum_b += neighbour_rgb.b;
    }
  }
  float n = (float)((1 + radius * 2) * (1 + radius * 2));
  *rgb = (RgbTriplet){.r = sum_r / n, .g = sum_g / n, .b = sum_b / n};
}

__global__ void sharpen_kernel(PpmImage image, size_t image_size, float threshold, float sharpen_factor, size_t m) {
    size_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (global_idx >= image_size)
        return;
    size_t x = global_idx % image.width;
    size_t y = global_idx / image.width;
    RgbTriplet rgb, blur, new_rgb;
    rgb = image.color_values_read[global_idx];
    blur_at(&image, m, x, y, &blur);
    if (rgb.r <= threshold)
      new_rgb = blur;
    else
      new_rgb = (RgbTriplet){
          .r = rgb.r + sharpen_factor * (rgb.r - blur.r),
          .g = rgb.g + sharpen_factor * (rgb.g - blur.g),
          .b = rgb.b + sharpen_factor * (rgb.b - blur.b)};
    new_rgb.r = (new_rgb.r >= 1.0f) ? 1.0f : ((new_rgb.r <= 0.0f) ? 0.0f : new_rgb.r);
    new_rgb.g = (new_rgb.g >= 1.0f) ? 1.0f : ((new_rgb.g <= 0.0f) ? 0.0f : new_rgb.g);
    new_rgb.b = (new_rgb.b >= 1.0f) ? 1.0f : ((new_rgb.b <= 0.0f) ? 0.0f : new_rgb.b);
    image.color_values_write[global_idx] = new_rgb;
}

__global__ void grayscale_kernel(PpmImage image, size_t image_size) {
    size_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (global_idx >= image_size)
        return;
    RgbTriplet rgb;
    rgb = image.color_values_read[global_idx];
    float y = 0.299f * rgb.r + 0.587f * rgb.g + 0.114f * rgb.b;
    RgbTriplet grayscale_rgb = (RgbTriplet){.r = y, .g = y, .b = y};
    image.color_values_write[global_idx] = grayscale_rgb;
}

int filter_ppm_image(PpmImage *image, float threshold, float sharpen_factor,
                     size_t m, int threads_per_block) {
  int exit_code = 0;
  size_t image_size = image->width * image->height;
  PpmImage device_image = (PpmImage){
      .width = image->width,
      .height = image->height,
      .max_value = image->max_value,
      .color_values_write = NULL,
      .color_values_read = NULL,
      .needs_flushing = image->needs_flushing,
  };
  size_t blocks = image_size / threads_per_block;
  if (image_size % threads_per_block > 0)
    blocks++;
  ASSERT(image != NULL, "PPM image is null", filter_error);
  hipMalloc(&device_image.color_values_read, image_size * sizeof(RgbTriplet));
  hipMalloc(&device_image.color_values_write, image_size * sizeof(RgbTriplet));
  hipMemcpy(device_image.color_values_read, image->color_values_read, image_size * sizeof(RgbTriplet), hipMemcpyHostToDevice);
  sharpen_kernel<<<blocks, threads_per_block>>>(device_image, image_size, threshold, sharpen_factor, m);
  ASSERT(hipGetLastError() == hipSuccess, "Error while invoking CUDA kernel on the device", filter_error);
  hipDeviceSynchronize();
  hipMemcpy(device_image.color_values_read, device_image.color_values_write, image_size * sizeof(RgbTriplet), hipMemcpyDeviceToDevice);
  grayscale_kernel<<<blocks, threads_per_block>>>(device_image, image_size);
  ASSERT(hipGetLastError() == hipSuccess, "Error while invoking CUDA kernel on the device", filter_error);
  hipDeviceSynchronize();
  hipMemcpy(image->color_values_read, device_image.color_values_write, image_size * sizeof(RgbTriplet), hipMemcpyDeviceToHost);
  exit_code = 1;
filter_error:
  if (device_image.color_values_read != NULL)
    hipFree(device_image.color_values_read);
  if (device_image.color_values_write != NULL)
    hipFree(device_image.color_values_write);
  return exit_code;
}

// MAIN SECTION

int main(int argc, char **argv) {
  int exit_code = EXIT_FAILURE;
  PpmImage *image = NULL;
  FILE *output_file = NULL, *source_file = NULL;
  int threads_per_block = 1024;
  ASSERT(argc >= 6, "Missing arguments (min.: 5)", exit);
  // Reads the runtime parameters
  size_t m, raw_threshold;
  float sharpen_factor, threshold;
  ASSERT(sscanf(argv[3], "%lu", &m),
         "Error reading variable radius' `m` integer", exit);
  ASSERT(sscanf(argv[4], "%lu", &raw_threshold),
         "Error reading sharpen's `threshold` integer", exit);
  threshold = ((float)raw_threshold) / 255.0f;
  ASSERT(threshold >= 0.0f && threshold <= 1.0f,
         "Sharpen's `threshold` integer isn't inside 0..255 interval", exit);
  ASSERT(sscanf(argv[5], "%f", &sharpen_factor),
         "Error reading sharpen's `sharpen_factor` float", exit);
  ASSERT(sharpen_factor >= 0.0f && sharpen_factor <= 2.0f,
         "Sharpen's `sharpen_factor` float isn't inside 0..2 interval", exit);
  if (argc >= 7) {
      ASSERT(sscanf(argv[6], "%d", &threads_per_block),
             "Error reading `threads_per_block` integer multiplier", exit);
      threads_per_block *= 256;
      ASSERT(threads_per_block >= 1 && threads_per_block <= 1024, "Filter's "
          "`threads_per_block` is multiplied by 256, it should be inside 1..4",
          exit);
  }
  // Tries to open/close the output file in append-mode just to test if it's possible
  output_file = fopen(argv[2], "a");
  ASSERT(output_file != NULL, "Error opening the output file", exit);
  ASSERT(fclose(output_file) == 0, "Error closing the output file", exit);
  output_file = NULL;
  // Opens the source file and reads the PPM image
  source_file = fopen(argv[1], "r");
  ASSERT(source_file != NULL, "Error opening the source file", exit);
  image = read_ppm_image(source_file);
  ASSERT(fclose(source_file) == 0, "Error closing the source file", exit);
  source_file = NULL;
  ASSERT(image != NULL, "Error reading the PPM image", exit);
  // Apply the PPM image filter
  ASSERT(filter_ppm_image(image, threshold, sharpen_factor, m, threads_per_block),
         "Error applying the filter to the PPM image", exit);
  // Saves the PPM image to the output file
  output_file = fopen(argv[2], "w");
  ASSERT(output_file != NULL, "Error opening the output file", exit);
  ASSERT(save_ppm_image(image, output_file), "Error saving the PPM image",
         exit);
  ASSERT(fclose(output_file) == 0, "Error closing the output file", exit);
  output_file = NULL;
  exit_code = EXIT_SUCCESS;
exit:
  free_ppm_image(&image);
  return exit_code;
}
