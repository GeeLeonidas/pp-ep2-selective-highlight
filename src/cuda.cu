#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2025 Guilherme Leoi <leoi.guilherme@aluno.ufabc.edu.br>
//
// SPDX-License-Identifier: AGPL-3.0-only

#include "filter.h"
#include "ppm.h"
#include <stddef.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

// PPM SECTION

#define ASSERT(expr, msg, exit_label)                                          \
  if (!(expr)) {                                                               \
    puts(msg);                                                                 \
    goto exit_label;                                                           \
  }
#define MAX_LINE 4096

PpmImage *read_ppm_image(FILE *source_file) {
  PpmImage *image = NULL;
  size_t image_size = 0;
  ASSERT(source_file != NULL, "Source file is NULL", read_ppm_image_error);
  image = (PpmImage*)malloc(sizeof(PpmImage));
  char header[2];
  ASSERT(fscanf(source_file, "%c%c", &header[0], &header[1]),
         "Error reading the file header", read_ppm_image_error);
  ASSERT(header[0] == 'P' && header[1] == '3',
         "Unsupported format (expected `P3`)", read_ppm_image_error)
  char line[MAX_LINE];
  do {
    ASSERT(fgets(line, MAX_LINE, source_file),
           "Error reading line(s) after header", read_ppm_image_error);
  } while (line[0] == '#' || line[0] == '\n');
  ASSERT(sscanf(line, "%lu %lu", &image->width, &image->height),
         "Error reading `width` and `height` integers", read_ppm_image_error);
  ASSERT(fscanf(source_file, "%hu", &image->max_value),
         "Error reading `max_value` integer", read_ppm_image_error);
  image_size = image->width * image->height;
  image->color_values_write = (RgbTriplet*)malloc(image_size * sizeof(RgbTriplet));
  image->color_values_read = (RgbTriplet*)malloc(image_size * sizeof(RgbTriplet));
  image->needs_flushing = 0;
  for (size_t idx = 0; idx < image_size; idx++) {
    uint16_t red, green, blue;
    ASSERT(fscanf(source_file, "%hu %hu %hu", &red, &green, &blue),
           "Error reading `red`, `blue` and `green` integers",
           read_ppm_image_error);
    RgbTriplet rgb =
        (RgbTriplet){.r = ((float)red) / ((float)image->max_value),
                     .g = ((float)green) / ((float)image->max_value),
                     .b = ((float)blue) / ((float)image->max_value)};
    write_at_idx_ppm_image(image, idx, rgb);
  }
  ASSERT(flush_ppm_image(image), "Error flushing the image write buffer",
         read_ppm_image_error);
  return image;
read_ppm_image_error:
  free_ppm_image(&image);
  return NULL;
}

int write_at_idx_ppm_image(PpmImage *image, size_t idx, RgbTriplet rgb) {
  ASSERT(image != NULL, "PPM image is NULL", write_at_idx_ppm_image_error);
  ASSERT(image->color_values_write != NULL, "PPM image write buffer is NULL",
         write_at_idx_ppm_image_error);
  ASSERT(idx < (image->width * image->height),
         "Error writing at out of bounds index from PPM image",
         write_at_idx_ppm_image_error);
  image->color_values_write[idx] = rgb;
  image->needs_flushing = 1;
  return 1;
write_at_idx_ppm_image_error:
  return 0;
}

int write_at_xy_ppm_image(PpmImage *image, size_t x, size_t y, RgbTriplet rgb) {
  ASSERT(image != NULL, "PPM image is NULL", write_at_xy_ppm_image_error);
  return write_at_idx_ppm_image(image, x + y * image->width, rgb);
write_at_xy_ppm_image_error:
  return 0;
}

int flush_ppm_image(PpmImage *image) {
  ASSERT(image != NULL, "PPM image is NULL", flush_ppm_image_error);
  ASSERT(image->color_values_read != NULL, "PPM image read buffer is NULL",
         flush_ppm_image_error);
  ASSERT(image->color_values_write != NULL, "PPM image write buffer is NULL",
         flush_ppm_image_error);
  if (image->needs_flushing) {
    size_t image_size = image->width * image->height;
    for (size_t idx = 0; idx < image_size; idx++)
      image->color_values_read[idx] = image->color_values_write[idx];
    image->needs_flushing = 0;
  }
  return 1;
flush_ppm_image_error:
  return 0;
}

int read_at_idx_ppm_image(PpmImage *image, size_t idx, RgbTriplet *rgb) {
  ASSERT(image != NULL, "PPM image is NULL", read_at_idx_ppm_image_error);
  ASSERT(image->color_values_read != NULL, "PPM image read buffer is NULL",
         read_at_idx_ppm_image_error);
  ASSERT(rgb != NULL, "RgbTriplet is NULL", read_at_idx_ppm_image_error);
  ASSERT(idx < (image->width * image->height),
         "Error reading at out of bounds index from PPM image",
         read_at_idx_ppm_image_error);
  *rgb = image->color_values_read[idx];
  return 1;
read_at_idx_ppm_image_error:
  return 0;
}

int read_at_xy_ppm_image(PpmImage *image, size_t x, size_t y, RgbTriplet *rgb) {
  ASSERT(image != NULL, "PPM image is NULL", read_at_xy_ppm_image_error);
  return read_at_idx_ppm_image(image, x + y * image->width, rgb);
read_at_xy_ppm_image_error:
  return 0;
}

int save_ppm_image(PpmImage *image, FILE *output_file) {
  size_t image_size = 0;
  ASSERT(image != NULL, "PPM image is NULL", save_ppm_image_error);
  ASSERT(output_file != NULL, "Output file is NULL", save_ppm_image_error);
  ASSERT(fprintf(output_file, "P3\n"), "Error writing PPM image header",
         save_ppm_image_error);
  ASSERT(fprintf(output_file, "%lu %lu\n", image->width, image->height),
         "Error writing `width` and `height` integers", save_ppm_image_error);
  ASSERT(fprintf(output_file, "%hu\n", image->max_value),
         "Error writing `max_value` integer", save_ppm_image_error);
  image_size = image->width * image->height;
  for (size_t idx = 0; idx < image_size; idx++) {
    RgbTriplet rgb;
    ASSERT(read_at_idx_ppm_image(image, idx, &rgb),
           "Error reading at index from PPM image", save_ppm_image_error);
    uint16_t red, green, blue;
    red = (uint16_t)roundf(rgb.r * ((float)image->max_value));
    green = (uint16_t)roundf(rgb.g * ((float)image->max_value));
    blue = (uint16_t)roundf(rgb.b * ((float)image->max_value));
    ASSERT(fprintf(output_file, "%hu %hu %hu\n", red, green, blue),
           "Error writing `red`, `green` and `blue` integers",
           save_ppm_image_error);
  }
  return 1;
save_ppm_image_error:
  return 0;
}

void free_ppm_image(PpmImage **image) {
  if (image == NULL || *image == NULL)
    return;
  if ((*image)->color_values_write) {
    free((*image)->color_values_write);
    (*image)->color_values_write = NULL;
  }
  if ((*image)->color_values_read) {
    free((*image)->color_values_read);
    (*image)->color_values_read = NULL;
  }
  free(*image);
  *image = NULL;
}

// FILTER SECTION

int grayscale(PpmImage *image) {
  if (image == NULL)
    return 0;
  size_t image_size = image->width * image->height;
  for (size_t idx = 0; idx < image_size; idx++) {
    RgbTriplet rgb;
    if (!read_at_idx_ppm_image(image, idx, &rgb))
      return 0;
    float y = 0.299f * rgb.r + 0.587f * rgb.g + 0.114f * rgb.b;
    RgbTriplet grayscale_rgb = (RgbTriplet){.r = y, .g = y, .b = y};
    if (!write_at_idx_ppm_image(image, idx, grayscale_rgb))
      return 0;
  }
  if (!flush_ppm_image(image))
    return 0;
  return 1;
}

size_t r_pixel(PpmImage *image, size_t m, size_t x, size_t y) {
  if (image == NULL)
    return 0;
  if (x > image->width || y > image->height)
    return 0;
  RgbTriplet rgb;
  size_t idx = x + y * image->width;
  if (!read_at_idx_ppm_image(image, idx, &rgb))
    return 0;
  float sum = rgb.r + rgb.g + rgb.b;
  return (((size_t)(sum * 255)) % m) + 1;
}

int blur_at(PpmImage *image, size_t m, size_t x, size_t y, RgbTriplet *rgb) {
  if (image == NULL)
    return 0;
  if (x > image->width || y > image->height)
    return 0;
  size_t radius = r_pixel(image, m, x, y);
  if (radius == 0)
    return 0;
  float sum_r = 0.0f, sum_g = 0.0f, sum_b = 0.0f;
  for (size_t i = 0; i <= 2 * radius; i++) {
    for (size_t j = 0; j <= 2 * radius; j++) {
      size_t neighbour_x = x + i;
      if (neighbour_x < radius)
        neighbour_x = 0;
      else
        neighbour_x -= radius;
      size_t neighbour_y = y + j;
      if (neighbour_y < radius)
        neighbour_y = 0;
      else
        neighbour_y -= radius;
      if (neighbour_x >= image->width)
        neighbour_x = image->width - 1;
      if (neighbour_y >= image->height)
        neighbour_y = image->height - 1;
      size_t neighbour_idx = neighbour_x + neighbour_y * image->width;
      RgbTriplet neighbour_rgb;
      if (!read_at_idx_ppm_image(image, neighbour_idx, &neighbour_rgb))
        return 0;
      sum_r += neighbour_rgb.r;
      sum_g += neighbour_rgb.g;
      sum_b += neighbour_rgb.b;
    }
  }
  float n = (float)((1 + radius * 2) * (1 + radius * 2));
  *rgb = (RgbTriplet){.r = sum_r / n, .g = sum_g / n, .b = sum_b / n};
  return 1;
}

float clamp_zero_one(float input) {
  return (input >= 1.0f) ? 1.0f : ((input <= 0.0f) ? 0.0f : input);
}

int sharpen(PpmImage *image, float threshold, float sharpen_factor, size_t m) {
  if (image == NULL)
    return 0;
  for (size_t x = 0; x < image->width; x++) {
    for (size_t y = 0; y < image->height; y++) {
      RgbTriplet rgb, blur, new_rgb;
      if (!read_at_xy_ppm_image(image, x, y, &rgb))
        return 0;
      if (!blur_at(image, m, x, y, &blur))
        return 0;
      if (rgb.r <= threshold)
        new_rgb = blur;
      else
        new_rgb = (RgbTriplet){
            .r = clamp_zero_one(rgb.r + sharpen_factor * (rgb.r - blur.r)),
            .g = clamp_zero_one(rgb.g + sharpen_factor * (rgb.g - blur.g)),
            .b = clamp_zero_one(rgb.b + sharpen_factor * (rgb.b - blur.b))};
      if (!write_at_xy_ppm_image(image, x, y, new_rgb))
        return 0;
    }
  }
  if (!flush_ppm_image(image))
    return 0;
  return 1;
}

int filter_ppm_image(PpmImage *image, float threshold, float sharpen_factor,
                     size_t m) {
  if (image == NULL)
    return 0;
  if (!sharpen(image, threshold, sharpen_factor, m))
    return 0;
  if (!grayscale(image))
    return 0;
  return 1;
}

// MAIN SECTION

int main(int argc, char **argv) {
  int exit_code = EXIT_FAILURE;
  PpmImage *image = NULL;
  FILE *output_file = NULL, *source_file = NULL;
  ASSERT(argc >= 6, "Missing arguments (min.: 5)", exit);
  // Reads the runtime parameters
  size_t m, raw_threshold;
  float sharpen_factor, threshold;
  ASSERT(sscanf(argv[3], "%lu", &m),
         "Error reading variable radius' `m` integer", exit);
  ASSERT(sscanf(argv[4], "%lu", &raw_threshold),
         "Error reading sharpen's `threshold` integer", exit);
  threshold = ((float)raw_threshold) / 255.0f;
  ASSERT(threshold >= 0.0f && threshold <= 1.0f,
         "Sharpen's `threshold` integer isn't inside 0..255 interval", exit);
  ASSERT(sscanf(argv[5], "%f", &sharpen_factor),
         "Error reading sharpen's `sharpen_factor` float", exit);
  ASSERT(sharpen_factor >= 0.0f && sharpen_factor <= 2.0f,
         "Sharpen's `sharpen_factor` float isn't inside 0..2 interval", exit);
  // Tries to open/close the output file in append-mode just to test if it's possible
  output_file = fopen(argv[2], "a");
  ASSERT(output_file != NULL, "Error opening the output file", exit);
  ASSERT(fclose(output_file) == 0, "Error closing the output file", exit);
  output_file = NULL;
  // Opens the source file and reads the PPM image
  source_file = fopen(argv[1], "r");
  ASSERT(source_file != NULL, "Error opening the source file", exit);
  image = read_ppm_image(source_file);
  ASSERT(fclose(source_file) == 0, "Error closing the source file", exit);
  source_file = NULL;
  ASSERT(image != NULL, "Error reading the PPM image", exit);
  // Apply the PPM image filter
  ASSERT(filter_ppm_image(image, threshold, sharpen_factor, m),
         "Error applying the filter to the PPM image", exit);
  // Saves the PPM image to the output file
  output_file = fopen(argv[2], "w");
  ASSERT(output_file != NULL, "Error opening the output file", exit);
  ASSERT(save_ppm_image(image, output_file), "Error saving the PPM image",
         exit);
  ASSERT(fclose(output_file) == 0, "Error closing the output file", exit);
  output_file = NULL;
  exit_code = EXIT_SUCCESS;
exit:
  free_ppm_image(&image);
  return exit_code;
}
